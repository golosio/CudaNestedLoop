#include "hip/hip_runtime.h"
/*
Copyright (C) 2016 Bruno Golosio
This program is free software: you can redistribute it and/or modify
it under the terms of the GNU General Public License as published by
the Free Software Foundation, either version 3 of the License, or
(at your option) any later version.
This program is distributed in the hope that it will be useful,
but WITHOUT ANY WARRANTY; without even the implied warranty of
MERCHANTABILITY or FITNESS FOR A PARTICULAR PURPOSE.  See the
GNU General Public License for more details.
You should have received a copy of the GNU General Public License
along with this program.  If not, see <http://www.gnu.org/licenses/>.
*/

#include <stdio.h>
#include <stdlib.h>

#include <cub/device/device_reduce.cuh>
#include <cub/device/device_radix_sort.cuh>
//#include <hipcub/hipcub.hpp>

#include "cuda_error.h"
#include "nested_loop.h"

//////////////////////////////////////////////////////////////////////
// declare here the function called by the nested loop 
__device__ void NestedLoopFunction(int ix, int iy, int val);
//////////////////////////////////////////////////////////////////////

namespace NestedLoop
{
  #include "Ny_th.h"
  void *d_sort_storage_;
  size_t sort_storage_bytes_;
  void *d_reduce_storage_;
  size_t reduce_storage_bytes_;

  int Nx_max_;
  int *d_max_Ny_;
  int *d_sorted_Ny_;

  int *d_idx_;
  int *d_sorted_idx_;

  int block_dim_x_;
  int block_dim_y_;
  int frame_area_;
  float x_lim_;

#ifdef WITH_CUMUL_SUM
  PrefixScan prefix_scan_;
  uint *d_Ny_cumul_sum_;
#endif
   
}

//////////////////////////////////////////////////////////////////////
__global__ void SimpleNestedLoopKernel(int Nx, int *Ny)
{
  int ix = (blockIdx.x * blockDim.x) + threadIdx.x;
  int iy = (blockIdx.y * blockDim.y) + threadIdx.y;
  if (ix<Nx && iy<Ny[ix]) {
    NestedLoopFunction(ix, iy, 2);
  }
}

//////////////////////////////////////////////////////////////////////
__global__ void  ParallelInnerNestedLoopKernel(int ix, int Ny)
{
  int iy = threadIdx.x + blockIdx.x * blockDim.x;
  if (iy<Ny) {
    NestedLoopFunction(ix, iy, 2);
  }
}

//////////////////////////////////////////////////////////////////////
__global__ void  ParallelOuterNestedLoopKernel(int Nx, int *d_Ny)
{
  int ix = threadIdx.x + blockIdx.x * blockDim.x;
  if (ix<Nx) {
    for (int iy=0; iy<d_Ny[ix]; iy++) {
      NestedLoopFunction(ix, iy, 2);
    }
  }
}


//////////////////////////////////////////////////////////////////////
__global__ void Frame1DNestedLoopKernel(int ix0, int dim_x, int dim_y,
					int *sorted_idx, int *sorted_Ny)
{
  int array_idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (array_idx<dim_x*dim_y) {
    int ix = ix0 + array_idx % dim_x;
    int iy = array_idx / dim_x;
    if (iy<sorted_Ny[ix]) {
      // call here the function that should be called by the nested loop
      NestedLoopFunction(sorted_idx[ix], iy, 2);
    }
  }
}

//////////////////////////////////////////////////////////////////////
__global__ void Frame2DNestedLoopKernel(int ix0, int dim_x, int dim_y,
					int *sorted_idx, int *sorted_Ny)
{
  int ix = (blockIdx.x * blockDim.x) + threadIdx.x;
  int iy = (blockIdx.y * blockDim.y) + threadIdx.y;
  if (ix<dim_x && iy<sorted_Ny[ix+ix0]) {
    // call here the function that should be called by the nested loop
    NestedLoopFunction(sorted_idx[ix+ix0], iy, 2);
  }
}

//////////////////////////////////////////////////////////////////////
__global__ void Smart1DNestedLoopKernel(int ix0, int iy0, int dim_x, int dim_y,
                                 int *sorted_idx, int *sorted_Ny)
{
  int array_idx = threadIdx.x + blockIdx.x * blockDim.x;
  if (array_idx<dim_x*dim_y) {
    int ix = ix0 + array_idx % dim_x;
    int iy = iy0 + array_idx / dim_x;
    if (iy<sorted_Ny[ix]) {
      // call here the function that should be called by the nested loop
      NestedLoopFunction(sorted_idx[ix], iy, 2);
    }
  }
}

//////////////////////////////////////////////////////////////////////
__global__ void Smart2DNestedLoopKernel(int ix0, int iy0, int dim_x,
					int dim_y, int *sorted_idx,
					int *sorted_Ny)
{
  int ix = (blockIdx.x * blockDim.x) + threadIdx.x;
  int iy = iy0 + (blockIdx.y * blockDim.y) + threadIdx.y;
  if (ix<dim_x && iy<sorted_Ny[ix+ix0]) {
    // call here the function that should be called by the nested loop
    NestedLoopFunction(sorted_idx[ix+ix0], iy, 2);
  }
}

#ifdef WITH_CUMUL_SUM
__device__ int locate(uint val, uint *data, int n)
{
  int i_left = 0;
  int i_right = n-1;
  int i = (i_left+i_right)/2;
  while(i_right-i_left>1) {
    if (data[i] > val) i_right = i;
    else if (data[i]<val) i_left = i;
    else break;
    i=(i_left+i_right)/2;
  }

  return i;
}

__global__ void CumulSumNestedLoopKernel(int Nx, uint *Ny_cumul_sum,
					 uint Ny_sum)
{
  uint blockId   = blockIdx.y * gridDim.x + blockIdx.x;
  uint array_idx = blockId * blockDim.x + threadIdx.x;
  if (array_idx<Ny_sum) {
    int ix = locate(array_idx, Ny_cumul_sum, Nx + 1);
    int iy = (int)(array_idx - Ny_cumul_sum[ix]);
    NestedLoopFunction(ix, iy, 2);
  }
}
#endif

//////////////////////////////////////////////////////////////////////
int NestedLoop::Init()
{
  return Init(65536*1024);
}

//////////////////////////////////////////////////////////////////////
int NestedLoop::Init(int Nx_max)
{
  if (Nx_max <= 0) return 0;

  block_dim_x_ = 32;
  block_dim_y_ = 32;
  frame_area_ = 65536*64;
  x_lim_ = 0.75;
  Nx_max_ = Nx_max;

  CudaSafeCall(hipMalloc(&d_max_Ny_, sizeof(int)));  
  CudaSafeCall(hipMalloc(&d_sorted_Ny_, Nx_max*sizeof(int)));
  CudaSafeCall(hipMalloc(&d_idx_, Nx_max*sizeof(int)));
  CudaSafeCall(hipMalloc(&d_sorted_idx_, Nx_max*sizeof(int)));

  int *h_idx = new int[Nx_max];
  for(int i=0; i<Nx_max; i++) {
    h_idx[i] = i;
  }  
  CudaSafeCall(hipMemcpy(d_idx_, h_idx, Nx_max*sizeof(int),
			  hipMemcpyHostToDevice));
  delete[] h_idx;
    
  // Determine temporary storage requirements for RadixSort
  d_sort_storage_ = NULL;
  sort_storage_bytes_ = 0;
  hipcub::DeviceRadixSort::SortPairs(d_sort_storage_, sort_storage_bytes_,
				  d_sorted_Ny_, d_sorted_Ny_, d_idx_,
				  d_sorted_idx_, Nx_max);
  // Determine temporary device storage requirements for Reduce
  d_reduce_storage_ = NULL;
  reduce_storage_bytes_ = 0;
  int *d_Ny = NULL;
  hipcub::DeviceReduce::Max(d_reduce_storage_, reduce_storage_bytes_, d_Ny,
			 d_max_Ny_, Nx_max);

  // Allocate temporary storage
  CudaSafeCall(hipMalloc(&d_sort_storage_, sort_storage_bytes_));
  CudaSafeCall(hipMalloc(&d_reduce_storage_, reduce_storage_bytes_));

#ifdef WITH_CUMUL_SUM
  prefix_scan_.Init();
  CudaSafeCall(hipMalloc(&d_Ny_cumul_sum_,
			  PrefixScan::AllocSize*sizeof(uint)));
#endif
  
  return 0;
}

//////////////////////////////////////////////////////////////////////
int NestedLoop::Run(int Nx, int *d_Ny)
{
  return Smart2DNestedLoop(Nx, d_Ny);
}

//////////////////////////////////////////////////////////////////////
int NestedLoop::SimpleNestedLoop(int Nx, int *d_Ny)
{
  // Find max value of Ny
  hipcub::DeviceReduce::Max(d_reduce_storage_, reduce_storage_bytes_, d_Ny,
			 d_max_Ny_, Nx);
  int max_Ny;
  CudaSafeCall(hipMemcpy(&max_Ny, d_max_Ny_, sizeof(int),
			  hipMemcpyDeviceToHost));
  return SimpleNestedLoop(Nx, d_Ny, max_Ny);
}

//////////////////////////////////////////////////////////////////////
int NestedLoop::SimpleNestedLoop(int Nx, int *d_Ny, int max_Ny)
{
  if (max_Ny < 1) max_Ny = 1;
  dim3 threadsPerBlock(block_dim_x_, block_dim_y_);  // block size
  dim3 numBlocks((Nx - 1)/threadsPerBlock.x + 1,
		 (max_Ny - 1)/threadsPerBlock.y + 1);
  SimpleNestedLoopKernel <<<numBlocks,threadsPerBlock>>>(Nx, d_Ny);
  hipDeviceSynchronize();
  CudaCheckError();
  
  return 0;
}

//////////////////////////////////////////////////////////////////////
int NestedLoop::ParallelInnerNestedLoop(int Nx, int *d_Ny)
{
  for (int ix=0; ix<Nx; ix++) {
    int Ny;
    CudaSafeCall(hipMemcpy(&Ny, &d_Ny[ix], sizeof(int),
			    hipMemcpyDeviceToHost));
    ParallelInnerNestedLoopKernel<<<(Ny+1023)/1024, 1024>>>(ix, Ny);
    // CudaCheckError(); // uncomment only for debugging
  }
  hipDeviceSynchronize();
  CudaCheckError();
  
  return 0;
}

//////////////////////////////////////////////////////////////////////
int NestedLoop::ParallelOuterNestedLoop(int Nx, int *d_Ny)
{
  ParallelOuterNestedLoopKernel<<<(Nx+1023)/1024, 1024>>>(Nx, d_Ny);
  hipDeviceSynchronize();
  CudaCheckError();
  
  return 0;
}

//////////////////////////////////////////////////////////////////////
int NestedLoop::Frame1DNestedLoop(int Nx, int *d_Ny)
{
  if (Nx <= 0) return 0;
  int dim_x, dim_y;

  // Run sorting operation
  hipcub::DeviceRadixSort::SortPairs(d_sort_storage_, sort_storage_bytes_,
				  d_Ny, d_sorted_Ny_, d_idx_, d_sorted_idx_,
				  Nx);
  
  int ix0 = Nx;
  while(ix0>0) {
    CudaSafeCall(hipMemcpy(&dim_y, &d_sorted_Ny_[ix0-1], sizeof(int),
			    hipMemcpyDeviceToHost));
    if (dim_y < 1) dim_y = 1;
    dim_x = (frame_area_ - 1) / dim_y + 1;
    ix0 -= dim_x;
    if (ix0<0) {
      dim_x += ix0;
      ix0 = 0;
    } 
    Frame1DNestedLoopKernel<<<(dim_x*dim_y+1023)/1024, 1024>>>
      (ix0, dim_x, dim_y, d_sorted_idx_, d_sorted_Ny_);
  }
  hipDeviceSynchronize();
  CudaCheckError();
  
  return 0;
}

//////////////////////////////////////////////////////////////////////
int NestedLoop::Frame2DNestedLoop(int Nx, int *d_Ny)
{
  if (Nx <= 0) return 0;
  // Sort the pairs (ix, Ny) with ix=0,..,Nx-1 in ascending order of Ny.
  // After the sorting operation, d_sorted_idx_ are the reordered indexes ix
  // and d_sorted_Ny_ are the sorted values of Ny 
  hipcub::DeviceRadixSort::SortPairs(d_sort_storage_, sort_storage_bytes_,
				  d_Ny, d_sorted_Ny_, d_idx_, d_sorted_idx_,
				  Nx);  
  int ix0 = Nx;	      // proceeds from right to left
  while(ix0>0) {
    int dim_x, dim_y;  // width and height of the rectangular frame
    CudaSafeCall(hipMemcpy(&dim_y, &d_sorted_Ny_[ix0-1], sizeof(int),
			    hipMemcpyDeviceToHost));
    if (dim_y < 1) dim_y = 1;
    // frame_area_ is the fixed value of the the rectangular frame area
    dim_x = (frame_area_ - 1) / dim_y + 1; // width of the rectangular frame
    ix0 -= dim_x; // update the index value
    if (ix0<0) {
      dim_x += ix0;  // adjust the width if ix0<0 
      ix0 = 0;
    }    
    dim3 threadsPerBlock(block_dim_x_, block_dim_y_);  // block size
    dim3 numBlocks((dim_x - 1)/threadsPerBlock.x + 1,
		   (dim_y - 1)/threadsPerBlock.y + 1);
    // run a nested loop kernel on the rectangular frame
    Frame2DNestedLoopKernel <<<numBlocks,threadsPerBlock>>>
      (ix0, dim_x, dim_y, d_sorted_idx_, d_sorted_Ny_);

  }
  hipDeviceSynchronize();
  CudaCheckError();
  
  return 0;
}

//////////////////////////////////////////////////////////////////////
int NestedLoop::Smart1DNestedLoop(int Nx, int *d_Ny)
{
  // Find max value of Ny
  hipcub::DeviceReduce::Max(d_reduce_storage_, reduce_storage_bytes_, d_Ny,
			 d_max_Ny_, Nx);
  int max_Ny;
  CudaSafeCall(hipMemcpy(&max_Ny, d_max_Ny_, sizeof(int),
			  hipMemcpyDeviceToHost));
  if (Nx <= 0) return 0;
  float f_Nx = 2.0*log((float)Nx)-5;
  int i_Nx = (int)floor(f_Nx);
  int Ny_th;
  if (i_Nx<0) {
    Ny_th = Ny_th_arr_[0];
  }
  else if (i_Nx>=Ny_arr_size_-1) {
    Ny_th = Ny_th_arr_[Ny_arr_size_-1];
  }
  else {
    float t = f_Nx - (float)i_Nx;
    Ny_th = Ny_th_arr_[i_Nx]*(1.0 - t) + Ny_th_arr_[i_Nx+1]*t;
  }
  if (max_Ny<Ny_th) {
    return SimpleNestedLoop(Nx, d_Ny, max_Ny);
  }

  if(max_Ny < 1) max_Ny = 1;
  
  int dim_x, dim_y;

  // Run sorting operation
  hipcub::DeviceRadixSort::SortPairs(d_sort_storage_, sort_storage_bytes_,
				  d_Ny, d_sorted_Ny_, d_idx_, d_sorted_idx_,
				  Nx);
  // CudaCheckError(); // uncomment only for debugging
  
  int ix1 = (int)round(x_lim_*Nx);
  if (ix1==Nx) ix1 = Nx - 1;
  int Ny1;
  CudaSafeCall(hipMemcpy(&Ny1, &d_sorted_Ny_[ix1], sizeof(int),
			  hipMemcpyDeviceToHost));
  if(Ny1 < 1) Ny1 = 1;

  dim3 threadsPerBlock(block_dim_x_, block_dim_y_);  // block size
  int nbx = (Nx - 1)/threadsPerBlock.x + 1;
  int nby = (Ny1 - 1)/threadsPerBlock.y + 1;
  Ny1 = nby*threadsPerBlock.y;
  
  dim3 numBlocks(nbx, nby);
  SimpleNestedLoopKernel <<<numBlocks,threadsPerBlock>>>(Nx, d_Ny);
  //CudaCheckError(); // uncomment only for debugging
  
  int ix0 = Nx;
  while(ix0>ix1) {
    CudaSafeCall(hipMemcpy(&dim_y, &d_sorted_Ny_[ix0-1], sizeof(int),
			    hipMemcpyDeviceToHost));
    dim_y -= Ny1;
    if (dim_y<=0) break;
    dim_x = (frame_area_ - 1) / dim_y + 1;
    ix0 -= dim_x;
    if (ix0<ix1) {
      dim_x += ix0 - ix1;
      ix0 = ix1;
    } 
    Smart1DNestedLoopKernel<<<(dim_x*dim_y+1023)/1024, 1024>>>
      (ix0, Ny1, dim_x, dim_y, d_sorted_idx_, d_sorted_Ny_);
    //CudaCheckError(); // uncomment only for debugging
  }
  hipDeviceSynchronize();
  CudaCheckError();
  
  return 0;
}

//////////////////////////////////////////////////////////////////////
int NestedLoop::Smart2DNestedLoop(int Nx, int *d_Ny)
{
  // Find max value of Ny
  hipcub::DeviceReduce::Max(d_reduce_storage_, reduce_storage_bytes_, d_Ny,
			 d_max_Ny_, Nx);
  int max_Ny;
  CudaSafeCall(hipMemcpy(&max_Ny, d_max_Ny_, sizeof(int),
			  hipMemcpyDeviceToHost));
  if (Nx <= 0) return 0;
  float f_Nx = 2.0*log((float)Nx)-5;
  int i_Nx = (int)floor(f_Nx);
  int Ny_th;
  if (i_Nx<0) {
    Ny_th = Ny_th_arr_[0];
  }
  else if (i_Nx>=Ny_arr_size_-1) {
    Ny_th = Ny_th_arr_[Ny_arr_size_-1];
  }
  else {
    float t = f_Nx - (float)i_Nx;
    Ny_th = Ny_th_arr_[i_Nx]*(1.0 - t) + Ny_th_arr_[i_Nx+1]*t;
  }
  if (max_Ny<Ny_th) {
    return SimpleNestedLoop(Nx, d_Ny, max_Ny);
  }

  if(max_Ny < 1) max_Ny = 1;

  int dim_x, dim_y;

  // Run sorting operation
  hipcub::DeviceRadixSort::SortPairs(d_sort_storage_, sort_storage_bytes_,
				  d_Ny, d_sorted_Ny_, d_idx_, d_sorted_idx_,
				  Nx);
  // CudaCheckError(); // uncomment only for debugging
  
  int ix1 = (int)round(x_lim_*Nx);
  if (ix1==Nx) ix1 = Nx - 1;
  int Ny1;
  CudaSafeCall(hipMemcpy(&Ny1, &d_sorted_Ny_[ix1], sizeof(int),
			  hipMemcpyDeviceToHost));
  if(Ny1 < 1) Ny1 = 1;

  dim3 threadsPerBlock(block_dim_x_, block_dim_y_);  // block size
  int nbx = (Nx - 1)/threadsPerBlock.x + 1;
  int nby = (Ny1 - 1)/threadsPerBlock.y + 1;
  Ny1 = nby*threadsPerBlock.y;
  
  dim3 numBlocks(nbx, nby);
  SimpleNestedLoopKernel <<<numBlocks,threadsPerBlock>>>(Nx, d_Ny);
  //CudaCheckError(); // uncomment only for debugging
  
  int ix0 = Nx;
  while(ix0>ix1) {
    CudaSafeCall(hipMemcpy(&dim_y, &d_sorted_Ny_[ix0-1], sizeof(int),
			    hipMemcpyDeviceToHost));
    dim_y -= Ny1;
    if (dim_y<=0) break;
    dim_x = (frame_area_ - 1) / dim_y + 1;
    ix0 -= dim_x;
    if (ix0<ix1) {
      dim_x += ix0 - ix1;
      ix0 = ix1;
    }

    dim3 threadsPerBlock(block_dim_x_, block_dim_y_);  // block size
    dim3 numBlocks((dim_x - 1)/threadsPerBlock.x + 1,
		   (dim_y - 1)/threadsPerBlock.y + 1);
    Smart2DNestedLoopKernel <<<numBlocks,threadsPerBlock>>>
      (ix0, Ny1, dim_x, dim_y, d_sorted_idx_, d_sorted_Ny_);
    //CudaCheckError(); // uncomment only for debugging      
  }
  hipDeviceSynchronize();
  CudaCheckError();
  
  return 0;
}

//////////////////////////////////////////////////////////////////////
#ifdef WITH_CUMUL_SUM
int NestedLoop::CumulSumNestedLoop(int Nx, int *d_Ny)
{
  prefix_scan_.Scan(d_Ny_cumul_sum_, (uint*)d_Ny, Nx);
  uint Ny_sum;
  CudaSafeCall(hipMemcpy(&Ny_sum, &d_Ny_cumul_sum_[Nx],
			  sizeof(uint), hipMemcpyDeviceToHost));
  //printf("Ny_sum %u\n", Ny_sum);
  if(Ny_sum>0) {
    uint grid_dim_x, grid_dim_y;
    if (Ny_sum<65536*1024) { // max grid dim * max block dim
      grid_dim_x = (Ny_sum+1023)/1024;
      grid_dim_y = 1;
    }
    else {
      grid_dim_x = 64; // I think it's not necessary to increase it
      if (Ny_sum>grid_dim_x*1024*65535) {
	printf("Ny sum %d larger than threshold %d\n",
	       Ny_sum, grid_dim_x*1024*65535);
	exit(-1);
      }
      grid_dim_y = (Ny_sum + grid_dim_x*1024 -1) / (grid_dim_x*1024);
    }
    dim3 numBlocks(grid_dim_x, grid_dim_y);
    CumulSumNestedLoopKernel<<<numBlocks, 1024>>>(Nx, d_Ny_cumul_sum_, Ny_sum);

    hipDeviceSynchronize();
    CudaCheckError();
  }
    
  return 0;
}
#endif
